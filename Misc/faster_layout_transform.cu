#include "hip/hip_runtime.h"
/*
Playing around with implementing layout transform / transposition efficiently
*/
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define COPY_LOADS_PER_THREAD 1
__global__ void CudaBaselineCopy(float *mat1, float *mat2) {
  int start_index_t0 = blockIdx.x * blockDim.x * COPY_LOADS_PER_THREAD;
  for (int i = 0; i < COPY_LOADS_PER_THREAD; i++) {
    int index_t0 = start_index_t0 + i * blockDim.x;
    mat2[index_t0 + threadIdx.x] = mat1[index_t0 + threadIdx.x];
  }
}

void CudaBaselineCopyWrapped(float *d_mat1, float *d_mat2, int rows, int cols,
                             int num_threads_per_block = 32) {
  int num_blocks = (rows * cols) / (COPY_LOADS_PER_THREAD * num_threads_per_block);
  CudaBaselineCopy<<<num_blocks, num_threads_per_block>>>(
      d_mat1, d_mat2);
}

__global__ void CudaTransposeNaive(float *mat, float *matT, int rows, int cols,
                                   int loads_per_thread,
                                   int num_threads_per_block) {
  int y_start = blockIdx.y * loads_per_thread;
  int x_start = blockIdx.x * num_threads_per_block + threadIdx.x;
  for (int i = 0; i < loads_per_thread; i++) {
    matT[x_start * rows + (y_start + i)] = mat[(y_start + i) * cols + x_start];
  }
}

void CudaTransposeNaiveWrapped(float *d_mat, float *d_matT, int rows, int cols,
                               int loads_per_thread = 32,
                               int num_threads_per_block = 32) {
  int num_blocks_y = rows / loads_per_thread;
  assert(rows % loads_per_thread == 0);
  int num_blocks_x = cols / num_threads_per_block;
  assert(cols % num_threads_per_block == 0);
  dim3 dimBlock(num_blocks_x, num_blocks_y, 1);
  dim3 dimGrid(num_threads_per_block, 1, 1);
  CudaTransposeNaive<<<dimBlock, dimGrid>>>(
      d_mat, d_matT, rows, cols, loads_per_thread, num_threads_per_block);
}

void doTransposeCUDA(float *h_mat, float *h_matT, int rows, int cols) {
  int matrix_size = rows * cols * sizeof(float);
  float *d_mat, *d_matT;

  hipMalloc((void **)&d_mat, matrix_size);
  hipMalloc((void **)&d_matT, matrix_size);
  hipMemcpy(d_mat, h_mat, matrix_size, hipMemcpyHostToDevice);
  hipMemcpy(d_matT, h_matT, matrix_size, hipMemcpyHostToDevice);

  // Call kernel
  CudaTransposeNaiveWrapped(d_mat, d_matT, rows, cols, 4, 1024);

  hipMemcpy(h_matT, d_matT, matrix_size, hipMemcpyDeviceToHost);
  hipFree(d_mat);
  hipFree(d_matT);
}

void doTransposeCPU(float *h_mat, float *h_matT, int rows, int cols) {
  for (int r = 0; r < rows; r++) {
    for (int c = 0; c < cols; c++) {
      h_matT[c * rows + r] = h_mat[r * cols + c];
    }
  }
}

void runTransposeExperiment(int matrix_rows, int matrix_cols) {
  float *h_mat = (float *)malloc(sizeof(float) * matrix_rows * matrix_cols);
  float *h_matT_experimental =
      (float *)malloc(sizeof(float) * matrix_rows * matrix_cols);
  float *h_matT_control =
      (float *)malloc(sizeof(float) * matrix_rows * matrix_cols);

  for (int i = 0; i < matrix_rows * matrix_cols; i++) {
    h_mat[i] = rand() % 100;
    h_matT_experimental[i] = 0;
    h_matT_control[i] = 0;
  }

  clock_t start, end;
  double cpu_time_used;

  start = clock();
  doTransposeCUDA(h_mat, h_matT_experimental, matrix_rows, matrix_cols);
  hipDeviceSynchronize();
  end = clock();
  cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
  printf("CUDA CPU time %.5f (s)\n", cpu_time_used);

  start = clock();
  doTransposeCPU(h_mat, h_matT_control, matrix_rows, matrix_cols);
  end = clock();
  cpu_time_used = ((double)(end - start)) / CLOCKS_PER_SEC;
  printf("CPU time %.5f (s)\n", cpu_time_used);

  for (int i = 0; i < matrix_rows * matrix_cols; i++) {
    assert(fabs(h_matT_experimental[i] - h_matT_control[i]) < 0.001);
  }

  free(h_mat);
  free(h_matT_experimental);
  free(h_matT_control);
}

void runCopyThroughputExperiment(int matrix_rows, int matrix_cols) {
  int matrix_size_bytes = sizeof(float) * matrix_cols * matrix_rows;
  float *h_mat1 = (float *)malloc(sizeof(float) * matrix_size_bytes);
  float *h_mat2 = (float *)malloc(sizeof(float) * matrix_size_bytes);

  for (int i = 0; i < matrix_rows * matrix_cols; i++) {
    h_mat1[i] = rand() % 100;
    h_mat2[i] = 1;
  }

  float *d_mat1, *d_mat2;
  hipMalloc((void **)&d_mat1, matrix_size_bytes);
  hipMalloc((void **)&d_mat2, matrix_size_bytes);
  hipMemcpy(d_mat1, h_mat1, matrix_size_bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_mat2, h_mat2, matrix_size_bytes, hipMemcpyHostToDevice);

  // Call kernel
  CudaBaselineCopyWrapped(d_mat1, d_mat2, matrix_rows, matrix_cols, 256);

  hipMemcpy(h_mat2, d_mat2, matrix_size_bytes, hipMemcpyDeviceToHost);
  hipFree(d_mat1);
  hipFree(d_mat2);

  for (int i = 0; i < matrix_rows * matrix_cols; i++) {
    assert(fabs(h_mat1[i] - h_mat2[i]) < 0.001);
  }
}

int main() {
  runCopyThroughputExperiment(1024, 1024);
  runTransposeExperiment(1024, 1024);
  return 0;
}